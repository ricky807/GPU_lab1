#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "timing.h"

typedef unsigned long long bignum;
//is prime function that both cpu and gpu use
__device__ __host__ int isPrime(bignum x){
    #ifndef __CUDA_ARCH__
        bignum i;
        bignum lim = (bignum)sqrt((float)x) + 1;

        if (x % 2 == 0)
        {
            return 0;
        }

        for (i = 3; i < lim; i += 2)
        {
            if (x % i == 0)
                return 0;
        }

        return 1;
    #else
        bignum i;
        bignum lim = (bignum) sqrt((double)x) + 1;
        if (x % 2 == 0){
            return 0;
        }
        for(i=2; i<lim; i++){
            if ( x % i == 0)
                return 0;
        }
        return 1;
    #endif
}
//GPU find prime function. This is probably where the problem is occuring
__global__ void findPrimes(int *results, int arr_size)
{
    // Get our global thread ID
    bignum index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < arr_size)
    {
        bignum number = 2 * index + 1;
        results[index] = isPrime(number);
    }
}

//CPU function. This is working perfectly
void computePrimes(double results[], bignum s, bignum n){
   
   bignum i;

   if(s % 2 == 0) s ++;  //make sure s is an odd number

   for(i=s; i< s+n; i = i + 2){
      results[i]=isPrime(i);
   }
}
//sums up primes. This also is working fine
int arrSum( double a[], bignum len )
{
    int i, s = 0;
    for( i = 0; i < len; i ++ )
        s += a[i];

    return s;
}

int main( int argc, char* argv[] )
{
    if (argc < 3)
    {
        printf("Usage: prime upbound\n");
        exit(-1);
    }
    // Get N and blockSize 
    bignum N = (bignum) atoi(argv[1]);
    bignum n = (bignum) atoi(argv[1]);
    bignum blockSize = atoi(argv[2]);
    int i;

    int *h_results;

    // Device input vectors
    int *d_results;
 
    // Size, in bytes, of each vector
    size_t bytes = (N+1)*sizeof(double);
    
    double now, then, scost, pcost;

    //this section takes care of the CPU computing. This sections works perfectly fine
    double *CPUArray;
    then = currentTime();
    CPUArray = (double*)malloc(bytes);
    for( i = 0; i < N + 1; i++ ) {
        CPUArray[i] = 0;
    }
    computePrimes(CPUArray, 0, N);
    now = currentTime();
    printf("Total number of primes in that range is: %d.\n", arrSum(CPUArray, N + 1));
    scost = now - then;
    printf("%%%%%% Serial code executiontime in second is %lf\n\n", scost);
    free(CPUArray);

    //this section is for GPU
    // Allocate memory for host
    size_t arr_size = (int)ceil((float) ((n - 1.0) / 2.0));
    size_t results_num_bytes = arr_size * sizeof(int);
    h_results = (int *)malloc(results_num_bytes);
    // Initialize vectors on host
    hipMalloc(&d_results, results_num_bytes);

    bignum a;
    // Initialize vectors on host
    for (a = 0; a < arr_size; a++)
    {
        h_results[a] = 0;
    }
    then = currentTime();
    hipMemcpy(d_results, h_results, results_num_bytes, hipMemcpyHostToDevice);
    
 
    int gridSize;
 
    // Number of thread blocks in grid
    gridSize = (int)ceil((float) ((n + 1.0) / 2.0 / blockSize));

    // Execute the Gpu function
    findPrimes<<<gridSize, blockSize>>>(d_results, arr_size);
 
    // Copy array back to host
    hipMemcpy(h_results, d_results, results_num_bytes, hipMemcpyDeviceToHost);
    bignum sum = 0;
    for (i = 0; i < n/2; i++){
        sum += h_results[i];
    }
    printf("Total number of primes in that range is: %lld.\n", sum);
    now = currentTime();
    pcost = now - then;
    printf("GPU execution time is : %lf\n\n", pcost);
    // Release device memory
    hipFree(d_results);
 
    // Release host memory
    free(h_results);
    printf("Speedup : %lf\n", scost - pcost);
    return 0;
}
